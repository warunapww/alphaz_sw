#include "hip/hip_runtime.h"
device int _atomicRegion = 0;
device int px = 0;
device int py = 0;
device int pz = 0;

bool isInAtomicRegion = false;
int pTileI,pTileJ,pTileK,pTileL;

if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
	while (atomicExch(&_atomicRegion, 1) != 0) {
	
	}
	    //critical region
	    if (pz != blockDim.z-1) {
	    	pTileK = atomicInc(&pz,blockDim.z);
	    	pTileJ = py;
	    	pTileI = px;
	    } else if (py != blockDim.y-1) {
	    	pTileK = 0;
	    	pz = 0;
	    	pTileJ = atomicInc(&py,blockDim.y);
	    	pTileI = px;
	    } else {
	    	pTileK = 0;
	    	pTileJ = 0;
	    	pz = 0;
	    	py = 0;
	    	pTileI = atomicInc(&px,blockDim.x);
	    }
	{
	    atomicExch(&_atomicRegion, 0);
    }
	
/*	while (!isInAtomicRegion) {
		if (atomicExch(&_atomicRegion, 1) == 0) {
		    //critical region
		    isInAtomicRegion = true;
		    atomicExch(&_atomicRegion, 0);
	    }
    }
*/    
} 